#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

const bool memSharing = false;
#define N (int)1e2

__global__ void vectAdd(int *a, int *b, int *c){

        int tID = blockIdx.x * blockDim.x + threadIdx.x;
        if(tID < N){
            c[tID] = a[tID] + b[tID];
        }
        __syncthreads();
    
    return;
}

int main(){
    // printf("Enter the number of vectors: ");
    // scanf("%d", &N);

    int *a, *b, *c;
    size_t n = N*sizeof(int);

    a = (int*)malloc(n);
    b = (int*)malloc(n);
    c = (int*)malloc(n);

    srand(time(NULL));
    srand(time(NULL));
    for(int i=0; i<N; i++){
        a[i] = rand()%100;
        b[i] = rand()%100;
    }

    // clock_t start_time, end_time;
    // printf("CPU code has started\n");
    // start_time = clock();
    // for(int i=0; i<N; i++){
    //     c[i] = a[i] + b[i];
    // }
    // end_time = clock();
    // printf("Time taken by CPU : %f\n", ((double)end_time-start_time)/CLOCKS_PER_SEC);

    printf("Preparing for GPU code\n");

    int *cudaA, *cudaB, *cudaC;

    hipMalloc(&cudaA, n);
    hipMalloc(&cudaB, n);
    hipMalloc(&cudaC, n);

    hipMemcpy(cudaA, a, n, hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, n, hipMemcpyHostToDevice);
    // cudaMemcpy(cudaC, c, n, cudaMemcpyHostToDevice);
    // cudaMemcpy(cudaC, c, n, cudaMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time_taken;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    int B = 10, T = N/B;
    printf("GPU code has started\n");
    hipEventRecord(start,0);

    vectAdd<<<B,T>>>(cudaA, cudaB, cudaC);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_taken, start, stop);
    printf("GPU code has ended\n");

    hipMemcpy(c, cudaC, n, hipMemcpyDeviceToHost);

    // for(int i=0; i<N; i++){
    //     printf("%d+%d=%d\n",a[i],b[i],c[i]);
    // }

    printf("%d+%d=%d\n",a[N-1],b[N-1],c[N-1]);

    printf("Time taken by GPU : %f\n", time_taken);

    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(cudaC);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
